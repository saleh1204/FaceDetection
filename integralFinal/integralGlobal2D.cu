#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define NUMBLOCK 1
#define BLOCKWIDTH 16  
#define NUMTHREAD 4
#define ASIZE 4

void printArray(int * image){
	int i,j;
	for (i = 0; i < ASIZE; ++i)
	{
		for (j = 0; j < ASIZE; ++j)
		{
			printf("%d\t", image[i * ASIZE + j]);
		}
		printf("\n");
	}
	printf("\n\n");
}

__global__ void prefixSum(int * img, int * integral)
{
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int i;

	printf("blockIdx = %d, blockDim = %d, threadIdx = %d, img[%d] = %d\n", blockIdx.x, blockDim.x, threadIdx.x, idx, img[idx]);
	printf("blockIdy = %d, blockDimy = %d, threadIdy = %d, img[%d][%d] = %d\n", blockIdx.y, blockDim.y, threadIdx.y, idx,idy, img[idx*ASIZE + idy]);



	//printf("blockIdx = %d, blockDim = %d, threadIdx = %d, img[%d] = %d\n", blockIdx.x, blockDim.x, threadIdx.x, idx, img[idx]);

	if (idy == 0)
	{
		integral[idx*ASIZE+idy] = img[idx*ASIZE+idy];
	}
	else
		integral[idx*ASIZE+idy] = img[idx*ASIZE+idy] + integral[idx*ASIZE+idy-1];

	printf("img[%d][%d] > %d, integral[] > %d\n", idx, idy,img[idx*ASIZE+idy], integral[idx*ASIZE+idy-1]);

	__syncthreads();
	
}

__global__ void columnSum(int * img, int * integral)
{
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int i;

	printf("idx > %d, idy > %d, img[] > %d, integral[] > %d\n", idx, idy, img[idx + idy*ASIZE], integral[idx + idy*ASIZE]);

	if (idx == 0)
		integral[idx + idy*ASIZE] = img[idx + idy*ASIZE];
	else
		integral[idx + idy*ASIZE] = img[idx + (idy*ASIZE)] + integral[idx + (idy-1)*ASIZE];

	__syncthreads();

}

int main()
{
//	const int SIZE = ASIZE;
	//int ASIZE = *(int *) argv[1];
	int *IMG_HOST, *INTG_HOST;
	int *IMG_DEV, *INTG_DEV;

	//Time initialization
	float timePassed;

	size_t size = ASIZE*sizeof(int);


	IMG_HOST = (int *)malloc(size*size);
	INTG_HOST = (int *)malloc(size*size);

	hipMalloc((void **) &IMG_DEV, size*size);
	hipMalloc((void **) &INTG_DEV, size*size);


	int i,j, random;
	for (i = 0; i < ASIZE; ++i)
	{
		//srand(i);
		for (j = 0; j < ASIZE; ++j)
		{
			//srand(j);
			IMG_HOST[i*ASIZE + j] = i*2 + j*4;
		}
	}
	
	printArray(IMG_HOST);
	dim3 grid(NUMBLOCK,NUMBLOCK), block(NUMTHREAD,NUMTHREAD);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(IMG_DEV, IMG_HOST, size*size, hipMemcpyHostToDevice);


	hipEventRecord(start, 0);

	prefixSum <<< grid, block >>> (IMG_DEV, INTG_DEV);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);


	hipEventElapsedTime(&timePassed, start,stop);
	printf("Time Spent Row: %0.5f\n", timePassed);


//#################################################################//
	
	hipMemcpy(INTG_HOST, INTG_DEV, size*size, hipMemcpyDeviceToHost);
	
	printArray(INTG_HOST);

	
	//hipMemcpy(INTG_DEV, INTG_HOST, size*size, hipMemcpyHostToDevice);
	

	//INTG_HOST = (int *)malloc(size*size);


	hipEventRecord(start, 0);

	columnSum <<< grid, block >>> (INTG_DEV, INTG_DEV);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&timePassed, start,stop);
	printf("Time Spent Column: %0.5f\n", timePassed);

	hipMemcpy(INTG_HOST, INTG_DEV, size*size, hipMemcpyDeviceToHost);

	printArray(INTG_HOST);

	//Free up the resources
	free(IMG_HOST);
	free(INTG_HOST);
	hipFree(IMG_DEV);
	hipFree(INTG_DEV);

	return 0;
}
