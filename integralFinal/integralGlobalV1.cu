#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define NUMBLOCK 1
#define BLOCKWIDTH 10
#define NUMTHREAD 1000
#define ASIZE 1000

void printArray(int * image){
	int i,j;
	for (i = 0; i < ASIZE; ++i)
	{
		for (j = 0; j < ASIZE; ++j)
		{
			printf("%d\t", image[i * ASIZE + j]);
		}
		printf("\n");
	}
	printf("\n\n");
}

__global__ void rowSum(int * img, int * integral)
{
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int i;


	printf("blockIdx = %d, blockDim = %d, threadIdx = %d, img[%d] = %d\n", blockIdx.x, blockDim.x, threadIdx.x, idx, img[idx]);

	//integral[idx] = img[idx];

	for(i = 0; i<ASIZE; i++){
		
		integral[idx*ASIZE+i] = img[idx*ASIZE+i] + img[idx*ASIZE+i+1];
		
		__syncthreads();
	}
	
	integral[idx*ASIZE+ASIZE-1] = img[idx*ASIZE+ASIZE-1];
}

__global__ void columnSum(int * img, int * integral)
{
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int i;

	//printf("blockIdx = %d, blockDim = %d, threadIdx = %d, img[%d] = %d\n", blockIdx.x, blockDim.x, threadIdx.x, idx, img[idx]);
	//printf("blockIdy = %d, blockDim = %d, threadIdy = %d, img[%d] = %d\n", blockIdx.y, blockDim.y, threadIdx.y, idy, img[idy]);

	//integral[idx] = img[idx];

	for(i = 0; i<ASIZE; i++){
		integral[idx + i*ASIZE] = img[idx+ (i*ASIZE)] + img[idx+ (i+1)*ASIZE];
		__syncthreads();
	}
	
	integral[idx+(ASIZE)*(ASIZE-1)] = img[idx+(ASIZE)*(ASIZE-1)];
}




int main()
{
	//int ASIZE = *(int *) argv[1];
	int *IMG_HOST, *INTG_HOST;
	int *IMG_DEV, *INTG_DEV;

	//Time initialization
	float timePassed;

	long size = ASIZE * sizeof(int);


	IMG_HOST = (int *)malloc(size*size);
	INTG_HOST = (int *)malloc(size*size);

	hipMalloc((void **) &IMG_DEV, size*size);
	hipMalloc((void **) &INTG_DEV, size*size);


	int i,j, random;
	for (i = 0; i < ASIZE; ++i)
	{
		//srand(i);
		for (j = 0; j < ASIZE; ++j)
		{
			//srand(j);
			IMG_HOST[i*ASIZE + j] = i*2 + j*4;
		}
	}
	
	//printArray(IMG_HOST);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(IMG_DEV, IMG_HOST, size*size, hipMemcpyHostToDevice);


	hipEventRecord(start, 0);

	rowSum <<< NUMTHREAD/BLOCKWIDTH, BLOCKWIDTH >>> (IMG_DEV, INTG_DEV);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&timePassed, start,stop);
	printf("Time Spent Row: %0.5f\n", timePassed);

	hipMemcpy(IMG_DEV, IMG_HOST, size*size, hipMemcpyHostToDevice);


	hipEventRecord(start, 0);

	columnSum <<< NUMTHREAD/BLOCKWIDTH, BLOCKWIDTH >>> (IMG_DEV, INTG_DEV);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&timePassed, start,stop);
	printf("Time Spent Row: %0.5f\n", timePassed);

	hipMemcpy(INTG_HOST, INTG_DEV, size*size, hipMemcpyDeviceToHost);

	//printArray(INTG_HOST);

	//Free up the resources
	free(IMG_HOST);
	free(INTG_HOST);
	hipFree(IMG_DEV);
	hipFree(INTG_DEV);

	return 0;
}
